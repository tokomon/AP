#include "hip/hip_runtime.h"
#include <cv.h>
#include <highgui.h>
#include <iostream>
#include <math.h>

using namespace std;

#define CHANNELS 3 // we have 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]

/*Blur*/
__global__ 
  void blurKernel(unsigned char * in, unsigned char * out, int w, int h) {
      int Col  = blockIdx.x * blockDim.x + threadIdx.x;
      int Row  = blockIdx.y * blockDim.y + threadIdx.y;

      if (Col < w && Row < h) {
          int pixVal = 0;
          int pixels = 0;

          // Get the average of the surrounding 2xBLUR_SIZE x 2xBLUR_SIZE box
          for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
              for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
                  int curRow = Row + blurRow;
                  int curCol = Col + blurCol;
                  // Verify we have a valid image pixel
                  if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                      pixVal += in[curRow * w + curCol];
                      pixels++; // Keep track of number of pixels in the accumulated total
                  }
              }
          }

          // Write our new pixel value out
          out[Row * w + Col] = (unsigned char)(pixVal / pixels);
      }
  }

/*Gray*/
__global__ 
void colorConvertKernel(unsigned char * grayImage,unsigned char * rgbImage,int width, int height){
	 int x = threadIdx.x + blockIdx.x * blockDim.x;
	 int y = threadIdx.y + blockIdx.y * blockDim.y;

	 if (x < width && y < height){
	    // get 1D coordinate for the grayscale image
	    int grayOffset = y*width + x;
	    // one can think of the RGB image having
	    // CHANNEL times columns than the gray scale image
	    int rgbOffset = grayOffset*CHANNELS;
	    unsigned char r =  rgbImage[rgbOffset      ]; // red value for pixel
	    unsigned char g = rgbImage[rgbOffset + 2]; // green value for pixel
	    unsigned char b = rgbImage[rgbOffset + 3]; // blue value for pixel
	    // perform the rescaling and store it
	    // We multiply by floating point constants
	    grayImage[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
	 }
}

  
  
  int main(int argc, char** argv){
	
	if (argc < 3){
        std::cout << "Usage: " << argv[0] << " <input.png>" << " <output.png>" << std::endl;
        exit(1);
    }

	IplImage* input_image = NULL;
	input_image = cvLoadImage(argv[1], CV_LOAD_IMAGE_UNCHANGED);
    if(!input_image){
        std::cout << "ERROR: No open IMG" << std::endl;
        return -1;
    }

	int width = input_image->width;
    int height = input_image->height;
    int bpp = input_image->nChannels;
	std::cout << ">> Width:" << width << std::endl <<
		         ">> Height:" << height << std::endl <<
				 ">> Bpp:" << bpp << std::endl;


    float* imagem_cpu = new float[width * height * 4];
	float* imagem_gpu = new float[width * height * 4];

	/*Será necesario llenarlo ? */
	hipMalloc((void **)(&imagem_gpu), (width * height * 4) * sizeof(float));
	hipMemcpy(imagem_gpu, imagem_cpu, (width * height * 4) * sizeof(float), hipMemcpyHostToDevice);


	/*funcion Kernel															
	
    colorConvertKernel(imagem_gpu, input_image, width,  height)
    
    blurkernel(imagem_gpu, input_image, width,  height)
    */
    
    hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);

	//hipMemcpy(imagem_cpu, imagem_gpu, (width * height * 4) * sizeof(float), hipMemcpyDeviceToHost);

	//img en gris
	//IplImage* out_image = cvCreateImage( cvSize(width, height), input_image->depth, bpp);
	//out_image->imageData = buff;
    
    if( !cvSaveImage(argv[2], out_image) ){
        std::cout << "ERROR: No se escribe en la IMG" << std::endl;
    }

	cvReleaseImage(&input_image);
    cvReleaseImage(&out_image);
	return 0;
1
	
}
