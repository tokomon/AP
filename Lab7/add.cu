#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float *A, float *B, float *C, int n){
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  //para saber que elemento se ejecuta
  if(i < n) C[i] = A[i] + B[i];
    //printf("%f ", C[i]);
}

void vecAdd(float* A, float* B, float* C, int n){
  int size = n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

  hipMalloc((void **) &d_C, size);
  vecAddKernel<<<ceil(n/256.0), 256>>>(d_A,d_B,d_C,n);
  //vecAddKernel(d_A,d_B,d_C,n);
  //ejecuta la funcion kernel
  //ceil funcion techo
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i;
  float *h_A,*h_B,*h_C;
  n=1000;
  //scanf("%d", &n);
  h_A = (float*) malloc(n*sizeof(float));
  h_B = (float*) malloc(n*sizeof(float));
  h_C = (float*) malloc(n*sizeof(float));
  for(i = 0; i < n; i++){
    //scanf("%f", &h_A[i]);
    h_A[i] = 11;
    h_B[i] = 11;
  //  printf("%f ", h_A[i]);

  //  h_C [i] = 11;
  }
  vecAdd(h_A,h_B,h_C,n);
cout<<" Suma "<<endl;
  for(i = 0; i < n; i++){
    printf("%f ", h_C[i]);
  }
  printf("\n");
  return 0;
}
