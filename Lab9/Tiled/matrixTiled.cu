

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define TILE_WIDTH 4

__global__
void matMultKernel(float *d_M, float *d_N, float *d_P, int Width){

  __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
 
  int Row = by*TILE_WIDTH + ty;
  int Col = bx*TILE_WIDTH + tx;

  float Pvalue = 0;
  int  m,k;
  for(m = 0; m < Width/TILE_WIDTH; ++m){
     Mds[ty][tx] = d_M[Row*Width+m*TILE_WIDTH + tx];
     Nds[ty][tx] = d_N[(m*TILE_WIDTH + ty) * Width + Col];
     __syncthreads();
     for(k = 0; k < TILE_WIDTH; ++k){
     	Pvalue += Mds[ty][k] * Nds[k][tx];
     }
     __syncthreads();
  }
  d_P[Row*Width + Col] = Pvalue;
}

void matMult(float* A, float* B, float* C, int n){
  int size = n*n*sizeof(float);
  float *d_A, *d_B, *d_C;

  hipMalloc((void **) &d_A, size);
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_B, size);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  hipMalloc((void **) &d_C, size);

  dim3 dimGrid(ceil(n/4.0),ceil(n/4.0),1);
  dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);
  matMultKernel<<<dimGrid, dimBlock>>>(d_A,d_B,d_C,n);
  
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

  hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main(){
  int n,i,j;
  float *h_A,*h_B,*h_C;
n=16;
  //scanf("%d", &n);
  h_A = (float*) malloc(n*n*sizeof(float));
  h_B = (float*) malloc(n*n*sizeof(float));
  h_C = (float*) malloc(n*n*sizeof(float));
  for(i = 0; i < n; i++){
    //scanf("%f", &h_A[i]);
    for(j = 0; j < n; j++)
      h_A[i*n+j] = i;
  }
  for(i = 0; i < n; i++){
    //scanf("%f", &h_B[i]);
    for(j = 0; j < n; j++)
      h_B[i*n+j] = 1;
  }
  matMult(h_A,h_B,h_C,n);
  for(i = 0; i < n; i++){
    for(j = 0; j < n; j++){
        printf("%f ", h_C[i*n+j]);
    }
    printf("\n");	
  }
  printf("\n");
  return 0;
}
