#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <stdlib.h>
#include "lodepng.h"
#include "lodepng.cu"

__global__
void gris(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  x = x*4;
  if(x < n*m*4) {
    unsigned char r = d_Pin[x];
    unsigned char g = d_Pin[x+1];
    unsigned char b = d_Pin[x+2];
    d_Pout[x] = 0.21f*r + 0.71f*g + 0.07f*b;
    d_Pout[x+1] = d_Pout[x];
    d_Pout[x+2] = d_Pout[x];
    d_Pout[x+3] = d_Pin[x+3];
  }
}

void dibujar(unsigned char* Pin, unsigned char* Pout, int n, int m){
  unsigned char* d_Pout, *d_Pin;
  long int size = n*m*4;
  hipMalloc((void **) &d_Pin,size);
  hipMemcpy(d_Pin, Pin, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_Pout,size);

  gris<<<(size-1)/256+1,256>>>(d_Pin,d_Pout,n,m);
  hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);
  hipFree(d_Pin); hipFree(d_Pout);
}

int main(){
  unsigned char *image, *out_image;
  int i;
  char name_in[100], name_out[100];
  unsigned width, height;

//ejemplo :
 /* name_in="Tokomon2.png";
  name_out="tokoBorrosa.png";*/
  scanf("%s %s", name_in, name_out);
  i = lodepng_decode32_file(&image, &width, &height, name_in);
  if(i < 0) printf("NO\n");
  out_image = (unsigned char*) malloc(width*height*4);

  dibujar(image,out_image,height,width);
  //dibujar out_imagen
  lodepng_encode32_file(name_out,out_image,width,height);

  free(image);
  free(out_image);
  return 0;
}
