#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <stdlib.h>
#include "lodepng.h"
#include "lodepng.cu"

__global__
void borrosa(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m){
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  
  int pixValR = 0, pixValG = 0, pixValB = 0;
  //BLUR_SIZE = nivel de intensidad
  int BLUR_SIZE = 20, blurRow, blurCol;

  if(x < n*m) {
    int pixels=0;
    for(blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1;++blurRow){
    	for(blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1;++blurCol){
	    int curX = blurCol + x;
	    int new_x = (blurRow*m + curX)*4;
	    if(curX > -1 && curX < n*m*4 && new_x > -1 && new_x < n*m*4){
	      pixValR += d_Pin[new_x];
	      pixValG += d_Pin[new_x+1];
	      pixValB += d_Pin[new_x+2];
	      pixels++;
	    }
	}

	d_Pout[x*4] = (unsigned char)(pixValR/pixels);
    	d_Pout[x*4+1] = (unsigned char)(pixValG/pixels);
    	d_Pout[x*4+2] = (unsigned char)(pixValB/pixels);
    	d_Pout[x*4+3] = d_Pin[x*4+3];
    }
  }
}

void dibujar(unsigned char* Pin, unsigned char* Pout, int n, int m){
  unsigned char* d_Pout, *d_Pin;
  long int size = n*m*4;
  hipMalloc((void **) &d_Pin,size);
  hipMemcpy(d_Pin, Pin, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &d_Pout,size);

  borrosa<<<(size-1)/256+1,256>>>(d_Pin,d_Pout,n,m);

  hipMemcpy(Pout, d_Pout, size, hipMemcpyDeviceToHost);
  hipFree(d_Pin); hipFree(d_Pout);
}

int main(){
  unsigned char *image, *out_image;
  int i;
  char name_in[100], name_out[100];
  unsigned width, height;
//ejemplo :
 /* name_in="Tokomon2.png";
  name_out="tokoBorrosa.png";*/
  scanf("%s %s", name_in, name_out);
  i = lodepng_decode32_file(&image, &width, &height, name_in);

  if(i < 0) printf("NO\n");
  out_image = (unsigned char*) malloc(width*height*4);

  dibujar(image,out_image,height,width);
  //dibujar out_imagen
  lodepng_encode32_file(name_out,out_image,width,height);

  free(image);
  free(out_image);
  return 0;
}
