/*Suma Matrices por Fila*/

#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__
void sumaMatrizKernel(float* A, float* B, float* C, int n)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if(i<n)
	{
		for(int j=0;j<n;j++)
			 C[i*n+j] = A[i*n+j] + B[i*n+j];

	}
	
}



void sumaMatrix(float* A, float* B, float* C, int tam)
{
	int size = (tam*tam) * sizeof(float);
	float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,size);
	hipMalloc((void**)&d_B,size);
	hipMalloc((void**)&d_C,size);

	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

	sumaMatrizKernel<<<ceil(tam/256.0),256>>>(d_A,d_B,d_C,tam);
	hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

	hipFree(d_A);hipFree(d_B);hipFree(d_C);
	
}

int main()
{
	int n = 10;
	float *h_A,*h_B,*h_C;
	h_A = new float[n*n];
	h_B = new float[n*n];
	h_C = new float[n*n];

	for(int i = 0; i < n; i++)
	{
	   for(int j = 0; j < n; j++)
	     h_A[i*n+j] = rand() % 100;
    }

    for(int i = 0; i < n; i++)
	{
	   for(int j = 0; j < n; j++)
	     h_B[i*n+j] = rand() % 100;
    }

	cout<<"Los vectores generados son: "<<endl;
    for(int i = 0; i < n; i++){
    	cout<<h_A[i]<<" ; ";
    }
	cout<<endl;
	for(int i = 0; i < n; i++){
	  cout<<h_B[i]<<" ; ";
	}
	cout<<endl;
    sumaMatrix(h_A,h_B,h_C,n);
    for(int i = 0; i < n; i++){
    	cout<<h_C[i]<<" ; ";
  	}

   cout<<endl;
   return 0;

	
}
